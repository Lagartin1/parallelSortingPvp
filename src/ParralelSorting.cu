#include <iostream>
#include <vector>
#include <random>
#include <algorithm>
#include <omp.h>
#include <hip/hip_runtime.h>



// Constantes para el Sort en GPU
#define BLOCK_SIZE 256
#define BUCKET_COUNT 8 // Número de buckets


using namespace std;
void generate_random_data(int* data, size_t n);
void merge(int* arr, int* left, int left_size, int* right, int right_size);
void parallel_merge_sort(int* arr, int n);
void cpu_parallel_sort(int* arr, size_t n);
__global__ void distribute_kernel(int* data, int* buckets, int* bucket_sizes, int n, int* pivots);
__global__ void local_sort_kernel(int* buckets, int* bucket_sizes, int bucket_idx, int bucket_capacity);
void gpu_sample_sort(int* data, size_t n);


// Función principal
int main(int argc, char** argv) {
    if (argc != 4) {
        cout << "Uso: ./prog <n> <modo> <nt>\n";
        cout << "  n: tamaño del array\n";
        cout << "  modo: 0 para CPU, 1 para GPU \n";
        cout << "  nt: número de threads (CPU)\n";
        return 1;
    }

    size_t n = stoull(argv[1]); // Tamaño del array
    int mode = stoi(argv[2]);  // Modo: CPU o GPU
    int num_threads = stoi(argv[3]); // Número de threads para CPU
    if (mode == 0 && num_threads <= 0) {
        cerr << "Error: número de threads inválido.\n";
        return 1;
    }
    if (mode != 0 && mode != 1) {
        cerr << "Error: modo inválido. Use 0 para CPU o 1 para GPU.\n";
        return 1;
    }

    vector<int> data(n); // Array de datos
    vector<int> verify_data(n); // Array para verificación
    generate_random_data(data.data(), n); // Generar datos aleatorios
    verify_data = data;

    double start_time, end_time;

    if (mode == 0) { // Modo CPU
        omp_set_num_threads(num_threads);
        start_time = omp_get_wtime();
        cpu_parallel_sort(data.data(), n);
        end_time = omp_get_wtime();
    } 
    else if (mode == 1) { // Modo GPU
        start_time = omp_get_wtime();
        gpu_sample_sort(data.data(), n);
        end_time = omp_get_wtime();
    }

    sort(verify_data.begin(), verify_data.end());
    bool is_correct = equal(data.begin(), data.end(), verify_data.begin());

    if (!is_correct) {
        cerr << "Error: El ordenamiento no es correcto!\n";
        return 1;
    }
    cout << "El ordenamiento es correcto!\n";
    cout << n << " elementos ordenados en " << end_time - start_time << " segundos.\n";

    return 0;
}

// Implementación de funciones

// Genera datos aleatorios en un array de enteros
void generate_random_data(int* data, size_t n) {
    random_device rd;
    mt19937 gen(rd());
    uniform_int_distribution<int> dist(0, INT_MAX);

    #pragma omp parallel for
    for (size_t i = 0; i < n; i++) {
        data[i] = dist(gen);
    }
}

// Mezcla dos subarrays ordenados en un array resultante
void merge(int* arr, int* left, int left_size, int* right, int right_size) {
    int i = 0, j = 0, k = 0;
    vector<int> temp(left_size + right_size);

    while (i < left_size && j < right_size) {
        if (left[i] <= right[j]) {
            temp[k++] = left[i++];
        } else {
            temp[k++] = right[j++];
        }
    }

    while (i < left_size) temp[k++] = left[i++];
    while (j < right_size) temp[k++] = right[j++];

    copy(temp.begin(), temp.end(), arr);
}

// Implementa Merge Sort en paralelo usando OpenMP
void parallel_merge_sort(int* arr, int n) {
    if (n <= 1) return;

    int mid = n / 2;

    #pragma omp task shared(arr) if(n > 100000)
    parallel_merge_sort(arr, mid);

    #pragma omp task shared(arr) if(n > 100000)
    parallel_merge_sort(arr + mid, n - mid);

    #pragma omp taskwait
    merge(arr, arr, mid, arr + mid, n - mid);
}

// Función principal para Merge Sort paralelo
void cpu_parallel_sort(int* arr, size_t n) {
    #pragma omp parallel
    {
        #pragma omp single
        parallel_merge_sort(arr, n);
    }
}
__global__ void distribute_kernel(int* data, int* buckets, int* bucket_sizes, int n, int* pivots) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n) {
        int val = data[tid];
        int bucket = 0;

        // Determinar en qué bucket cae el valor
        while (bucket < BUCKET_COUNT - 1 && val > pivots[bucket]) {
            bucket++;
        }

        // Incrementar el tamaño del bucket y agregar el valor
        int pos = atomicAdd(&bucket_sizes[bucket], 1);
        buckets[bucket * n + pos] = val;
    }
}

__global__ void local_sort_kernel(int* buckets, int* bucket_sizes, int bucket_idx, int bucket_capacity) {
    extern __shared__ int shared_data[]; // Memoria compartida
    int tid = threadIdx.x;

    // Tamaño del bucket actual
    int size = bucket_sizes[bucket_idx];
    if (tid >= size) return;

    // Copiar los datos del bucket a la memoria compartida
    int* bucket_data = buckets + bucket_idx * bucket_capacity;
    shared_data[tid] = bucket_data[tid];
    __syncthreads();

    // Variables para el stack explícito
    int left_stack[32];
    int right_stack[32];
    int stack_size = 0;

    // Inicializar el stack con los límites del arreglo
    if (tid == 0) {
        left_stack[stack_size] = 0;
        right_stack[stack_size] = size - 1;
        stack_size++;
    }
    __syncthreads();

    // Procesar el stack
    while (stack_size > 0) {
        int left, right;

        if (tid == 0) {
            // Extraer límites del stack
            stack_size--;
            left = left_stack[stack_size];
            right = right_stack[stack_size];
        }
        __syncthreads();

        // Elegir el pivote como el elemento del medio
        int pivot = shared_data[(left + right) / 2];

        // Particionar el arreglo
        int i = left;
        int j = right;

        while (i <= j) {
            while (shared_data[i] < pivot) i++;
            while (shared_data[j] > pivot) j--;

            if (i <= j) {
                // Intercambiar
                int temp = shared_data[i];
                shared_data[i] = shared_data[j];
                shared_data[j] = temp;
                i++;
                j--;
            }
        }
        __syncthreads();

        // Añadir nuevos subarreglos al stack
        if (tid == 0) {
            if (left < j) {
                left_stack[stack_size] = left;
                right_stack[stack_size] = j;
                stack_size++;
            }
            if (i < right) {
                left_stack[stack_size] = i;
                right_stack[stack_size] = right;
                stack_size++;
            }
        }
        __syncthreads();
    }

    // Copiar los datos ordenados de vuelta al bucket
    bucket_data[tid] = shared_data[tid];
}


void gpu_sample_sort(int* data, size_t n) {
    int* d_data, *d_buckets, *d_bucket_sizes, *d_pivots;
    int* pivots = new int[BUCKET_COUNT - 1];
    int bucket_capacity = n;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int num_sms = prop.multiProcessorCount;
    int num_blocks = num_sms * 5;
    // Generar pivotes (muestreo uniforme)
    for (int i = 0; i < BUCKET_COUNT - 1; i++) {
        pivots[i] = (i + 1) * (INT_MAX / BUCKET_COUNT);
    }

    // Reservar memoria en GPU
    hipMalloc(&d_data, n * sizeof(int));
    hipMalloc(&d_buckets, BUCKET_COUNT * bucket_capacity * sizeof(int));
    hipMalloc(&d_bucket_sizes, BUCKET_COUNT * sizeof(int));
    hipMalloc(&d_pivots, (BUCKET_COUNT - 1) * sizeof(int));

    // Copiar datos y pivotes al dispositivo
    hipMemcpy(d_data, data, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_pivots, pivots, (BUCKET_COUNT - 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_bucket_sizes, 0, BUCKET_COUNT * sizeof(int));

    // Paso 1: Distribuir elementos en buckets
    distribute_kernel<<<num_blocks, BLOCK_SIZE>>>(d_data, d_buckets, d_bucket_sizes, n, d_pivots);

    // Paso 2: Ordenar localmente cada bucket
    for (int i = 0; i < BUCKET_COUNT; i++) {
        int shared_memory_size = bucket_capacity * sizeof(int);
        local_sort_kernel<<<1, bucket_capacity, shared_memory_size>>>(d_buckets, d_bucket_sizes, i, bucket_capacity);
        hipDeviceSynchronize();
     }

    // Paso 3: Combinar buckets
    int offset = 0;
    for (int i = 0; i < BUCKET_COUNT; i++) {
        int size;
        hipMemcpy(&size, &d_bucket_sizes[i], sizeof(int), hipMemcpyDeviceToHost);

        hipMemcpy(data + offset, d_buckets + i * bucket_capacity, size * sizeof(int), hipMemcpyDeviceToHost);
        offset += size;
    }

    // Liberar memoria
    delete[] pivots;
    hipFree(d_data);
    hipFree(d_buckets);
    hipFree(d_bucket_sizes);
    hipFree(d_pivots);
}